#include "hip/hip_runtime.h"
#include <iostream>
#include <cassert>

//CUDA kernel for vector addition
//No change when using CUDA unified memory
__global__ void vectorAdd(int *a, int *b, int *c, int N) {
    //calculate global thread ID
    int tid = (blockDim.x * blockIdx.x) + threadIdx.x;

    //Boundary check
    if (tid < N) {
        c[tid] = a[tid] + b[tid];
    }
}

int main() {
    //Array size of 2^16 (65536 elements), bitshift left
    const int N = 1 << 16;
    size_t bytes = N * sizeof(int);

    //Declare unified memory pointers
    int *a, *b, *c;

    //Allocating memory for these pointer
    hipMallocManaged(&a, bytes);
    hipMallocManaged(&b, bytes);
    hipMallocManaged(&c, bytes);

    //Get the device ID for prefetching calls
    int id = hipGetDevice(&id);

    std::cout << "Device ids " << id << '\n';

    hipMemAdvise(a, bytes, hipMemAdviseSetPreferredLocation, hipCpuDeviceId);
    hipMemAdvise(b, bytes, hipMemAdviseSetPreferredLocation, hipCpuDeviceId);
    hipMemPrefetchAsync(c, bytes, id);

    //Initialize vectors
    for (int i = 0; i < N; i++) {
        a[i] = rand() % 100;
        b[i] = rand() % 100;
    }

    //Pre-fetch 'a' and 'b' arrays to the specified device (GPU)
    hipMemAdvise(a, bytes, hipMemAdviseSetReadMostly, id);
    hipMemAdvise(b, bytes, hipMemAdviseSetReadMostly, id);
    hipMemPrefetchAsync(a, bytes, id);
    hipMemPrefetchAsync(b, bytes, id);

    //Threads per CTA (1024 threads per CTA)
    int BLOCK_SIZE = 1 << 10;

    //CTA's per grid
    int GRID_SIZE = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;

    //Call CUDA kernel
    vectorAdd<<<GRID_SIZE, BLOCK_SIZE>>>(a, b, c, N);

    // wait for all previous operations before using values
    // We need this because we don't get the implicit synchronization
    // of hipMemcpy like in the origin example
    hipDeviceSynchronize();

    //Prefetch to the host (CPU)
    hipMemPrefetchAsync(a, bytes, hipCpuDeviceId);
    hipMemPrefetchAsync(b, bytes, hipCpuDeviceId);
    hipMemPrefetchAsync(c, bytes, hipCpuDeviceId);

    //Verify the result on the CPU
    for (int i = 0; i < N; i++) {
        assert(c[i] == a[i] + b[i]);
    }

    //Free unified memory (same as memory allocated with hipMalloc)
    hipFree(a);
    hipFree(b);
    hipFree(c);

   std::cout << "Completed successfully!\n";

   return 0;

}
